
/**
*   CS6023: GPU Programming 
*   Assignment 2
*   
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use cudaFree() 
*   to free up memory as soon as you're done with an allocation. 
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define ll long long 
using namespace std;

using std::cin;
using std::cout;


__global__ void solve(long long int *ans, long long int *gmat, long long int *gfilter, int m, int n, int k){

  extern __shared__ long long int filter[]; 
  unsigned tid = blockIdx.x * blockDim.x + threadIdx.x; 

  if ( tid < m * n){
    int row = tid/n; 
    int col = tid % n;
    int workperthread = ceil( (k*k)/(float)n); 

    for(int i = threadIdx.x; i < k*k; i += workperthread){
        filter[i] = gfilter[i]; 
    }

    __syncthreads(); 

    long long int sum = 0; 
    for(int i = row - k/2, r = 0; i < min(row + k/2 + 1, m); i++, r++){
        if ( i < 0 ) continue; 
        for(int j = col - k/2, c = 0; j < min (col + k/2 + 1, n); j++, c++){
            if ( j < 0) continue; 
            sum += filter[r*k + c] * gmat[i*n + j]; 
        }
    }
    ans[ row*n + col] = sum; 
  }

}

int main(int argc, char** argv) {

    int m, n, k; 
    cin >> m >> n >> k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];

    long int* h_ans = new long int[m * n];

    long long int *gmat, *gfilter, *ans; 

    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    hipMalloc(&gmat, (m*n)*sizeof(long long int));     
    hipMemcpy(gmat, h_mat, (m*n)*sizeof(long long int), hipMemcpyHostToDevice);     
    hipMalloc(&gfilter, (k*k)*sizeof(long long int)); 
    hipMemcpy(gfilter, h_filter, (k*k)*sizeof(long long int), hipMemcpyHostToDevice); 
    hipMalloc(&ans, (m*n)*sizeof(long long int));                 
    
    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    **/

    /****************************************************Start Here***********************************************************/
    
    int nblocks = ceil( (m*n)/ (float)1024); 
    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch
    solve <<< nblocks, 1024, (k*k)*sizeof(long long int) >>> (ans, gmat, gfilter, m, n, k); 
    hipDeviceSynchronize(); 
    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch
    
    hipFree(gmat); 
    hipFree(gfilter); 
    hipMemcpy(h_ans, ans, (m*n)*sizeof(long long int), hipMemcpyDeviceToHost);     
    hipFree(ans); 
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;    
    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */


    
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}