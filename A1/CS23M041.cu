/**
*   CS6023: GPU Programming 
*   Assignment 1
*   
*   Please don't change any existing code in this file.
*
*   You can add your code whereever needed. Please add necessary memory APIs
*   for your implementation. Use cudaFree() to free up memory as soon as you're
*   done with an allocation. This will ensure that you don't run out of memory 
*   while running large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;


__global__
void CalculateHadamardProduct(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    unsigned bid = blockIdx.x;
    unsigned tid = threadIdx.x;
    unsigned threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    unsigned i = (bid * threadsPerBlock + tid)/ N;
    unsigned j = (bid * threadsPerBlock + tid) % N;
    unsigned cur = i * N + j;
    if ( cur < N*N ){
        A[cur] = A[cur] * B[ j*N + i];
    }
}


__global__
void FindWeightMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    unsigned bid = blockIdx.x;
    unsigned tid = threadIdx.x * blockDim.x + threadIdx.y;
    unsigned threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    unsigned i = (bid * threadsPerBlock + tid)/ N;
    unsigned j = (bid * threadsPerBlock + tid) % N;
    unsigned cur = i * N + j;
    if ( cur < N*N && B[cur] > A[cur]){
          A[cur] = B[cur];
    }
}

__global__
void CalculateFinalMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    unsigned bid = blockIdx.x * gridDim.y + blockIdx.y;
    unsigned tid = threadIdx.x * blockDim.x + threadIdx.y;
    unsigned threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    unsigned i = (bid * threadsPerBlock + tid)/ ( 2* N);
    unsigned j = (bid * threadsPerBlock + tid) % ( 2* N);
    unsigned cur = i * 2 * N + j;
    if ( cur < 4*N*N){
        B[cur] *= A[ (i%N)*N + j%N];
    }
}

int main(int argc, char** argv) {


    int N;
    cin >> N;
    long int* A = new long int[N * N];
    long int* B = new long int[N * N];
    long int* C = new long int[N * N];
    long int* D = new long int[2 * N * 2 * N];


    for (long int i = 0; i < N * N; i++) {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++) {
        cin >> D[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    */

    long int* d_A;
    long int* d_B;
    long int* d_C;
    long int* d_D;

    hipMalloc(&d_A, N * N * sizeof(long int));
    hipMemcpy(d_A, A, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMalloc(&d_B, N * N * sizeof(long int));
    hipMemcpy(d_B, B, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMalloc(&d_C, N * N * sizeof(long int));
    hipMemcpy(d_C, C, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMalloc(&d_D, 4 * N * N * sizeof(long int));
    hipMemcpy(d_D, D, 4 * N * N * sizeof(long int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);


    auto start = std::chrono::high_resolution_clock::now();
    CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    hipDeviceSynchronize();    

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;
    hipFree(d_B); 

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);


    start = std::chrono::high_resolution_clock::now();
    FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);
    hipDeviceSynchronize();    
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;
    hipFree(d_C); 

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);


    start = std::chrono::high_resolution_clock::now();
    CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);
    hipDeviceSynchronize();    
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;
    hipFree(d_A); 
    // Make sure your final output from the device is stored in d_D.

    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);
    hipFree(d_D); 
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < 2 * N; i++) {
            for (long int j = 0; j < 2 * N; j++) {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}